#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA check error
#define cuda_check_error(msg) \
    { \
        hipError_t err = hipGetLastError(); \
        if( hipSuccess != err) { \
            fprintf(stderr, "[GPUJPEG] [Error] %s (line %i): %s: %s.\n", \
                __FILE__, __LINE__, msg, hipGetErrorString( err) ); \
            exit(-1); \
        } \
    } \

__global__
void get_value(int* index, int* value)
{
    int x[3];
    for ( int i = 0; i < 3; i++ )
        x[i] = 55;

    *value = x[*index];
}

int main()
{
    int* d_index;
    int* d_value;
    hipMalloc((void**)&d_index, sizeof(int));
    hipMalloc((void**)&d_value, sizeof(int));
    cuda_check_error("Alloc failed");

    int index = 0;
    int value = 0;
    hipMemcpy(d_index, &index, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_value, &value, sizeof(int), hipMemcpyHostToDevice);
    cuda_check_error("Init failed");

    get_value<<<1, 1>>>(d_index, d_value);
    hipDeviceSynchronize();
    cuda_check_error("Kernel failed");

    hipMemcpy(&index, d_index,  sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&value, d_value,  sizeof(int), hipMemcpyDeviceToHost);
    cuda_check_error("Copy failed");
    printf("index = %d\n", index);
    printf("value = %d\n", value);

    return 0;
}
